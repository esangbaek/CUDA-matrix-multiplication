
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define TILE_WIDTH 32

__global__ void MatrixMulKernelShared(float* d_M, float* d_N, float* d_P, int Md_row, int Md_col, int Nd_col)
{
	int Row = blockIdx.x * blockDim.x + threadIdx.x;
	int Col = blockIdx.y * blockDim.y + threadIdx.y;

	__shared__ float blockM[TILE_WIDTH][TILE_WIDTH];
	__shared__ float blockN[TILE_WIDTH][TILE_WIDTH];

	int block_row = threadIdx.x;
	int block_col = threadIdx.y;

	float block_sum = 0;

	for(int i = 0; i < ceil(Md_col/(TILE_WIDTH*1.0)); i++)
	{	
		
		if((Row < Md_row) && (Col < Nd_col))
		{
			blockM[block_row][block_col] = d_M[Row * Md_col + TILE_WIDTH * i + block_row];
			blockN[block_row][block_col] = d_N[(TILE_WIDTH * i + block_row)*Nd_col + Col];
		}
		__syncthreads();

		for(int j = 0; j < TILE_WIDTH; j++)
		{
			block_sum += blockM[block_row][j] * blockN[j][block_col];
		}
		__syncthreads();
	}

	d_P[Row * Nd_col + Col] = block_sum;
}

int main(int argc, const char **argv)
{
	int Md_row,Md_col, Nd_row,Nd_col;
	if(argc == 5)
	{
		Md_row = atoi(argv[1]);
		Md_col = atoi(argv[2]);
		Nd_row = atoi(argv[3]);
		Nd_col = atoi(argv[4]);
		if(Md_col != Nd_row)
		{
			printf("Invalid matrix size!\n");
			exit(0);
		}
	}else{
		printf("usage : %s <M row size> <M col size> <N row size> <N col size>\n",argv[0]);
		exit(0);
	}

	//Host
	float *Md, *Nd, *Pd;
	//Device
	float *d_M, *d_N, *d_P;

	hipEvent_t start, end;

	float time_ms=0;

	hipEventCreate(&start);
	hipEventCreate(&end);


	Md = (float*)malloc(Md_row * Md_col * sizeof(float));
	Nd = (float*)malloc(Nd_row * Nd_col * sizeof(float));
	Pd = (float*)malloc(Md_row * Nd_col * sizeof(float));
	for(int i=0;i<Md_row*Md_col;i++)
	{
		Md[i] = 1.0;
	}
	for(int j=0;j<Nd_row*Nd_col;j++)
	{
		Nd[j] = 1.0;
	}

	hipMalloc((void **)&d_M, Md_row * Md_col * sizeof(float));
	hipMalloc((void **)&d_N, Nd_row * Nd_col * sizeof(float));
	hipMalloc((void **)&d_P, Md_row * Nd_col * sizeof(float));
	hipMemset(d_M, 0, Md_row * Md_col * sizeof(float));
	hipMemset(d_N, 0, Nd_row * Nd_col * sizeof(float));
	hipMemset(d_P, 0, Md_row * Nd_col * sizeof(float));
	hipMemcpy(d_M, Md, Md_row * Md_col * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_N, Nd, Nd_row * Nd_col * sizeof(float), hipMemcpyHostToDevice);

	hipEventRecord(start, 0);

	//Kernel code
	dim3 dimGrid(ceil(Md_row/(TILE_WIDTH*1.0)), ceil(Nd_col/(TILE_WIDTH*1.0)),1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
	MatrixMulKernelShared<<< dimGrid, dimBlock >>>(d_M, d_N, d_P, Md_row, Md_col, Nd_col);

	hipEventRecord(end, 0);

	hipEventSynchronize(end);
	hipEventElapsedTime(&time_ms, start, end);
	hipDeviceSynchronize();

	hipMemcpy(Pd, d_P, Md_row * Nd_col * sizeof(float), hipMemcpyDeviceToHost);

	printf("Accesses to Shared Memory\n");
	printf("TILE WIDTH : %d\n", TILE_WIDTH);
	printf("(%d x %d),(%d x %d) Matrix\n\n",Md_row, Md_col, Nd_row, Nd_col);	
	printf("Execution time for kernel : %f ms\n", time_ms);
	
	float result = (float)Md_col;
	
	for(int i=0;i<Md_row;i++)
	{
		for(int j=0;j<Nd_col;j++)
		{
			if(result!=Pd[i*Nd_col+j]){
				printf("Wrong answer\n");
				goto quit;
			}
		}
	}
	
quit:
	hipFree(d_M);
	hipFree(d_N);
	hipFree(d_P);
	free(Md);
	free(Nd);
	free(Pd);

	return 0;
}
